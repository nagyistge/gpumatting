#include "hip/hip_runtime.h"
/*
 * Matting.cu is part of gpumatting and Copyright Philip G. Lee <rocketman768@gmail.com> 2013
 * all rights reserved.
 */

#include <hip/hip_runtime.h>
#include "BandedMatrix.h"
#include "Vector.cu"

/*!
 * \brief Generate Levin's Laplacian for a given image.
 * 
 * Needs to be called with 2D blocks, enough for 1 thread/pixel
 *
 * NOTE: requires \c levinLaplacian_image to be a globally visible
 *       texture<float4,hipTextureType2D,hipReadModeElementType>
 * NOTE: requires \c levinLaplacian_trimap to be a globally visible
 *       texture<float,hipTextureType2D,hipReadModeElementType>
 * 
 * \param L output banded sparse Laplacian
 * \param b output right-hand side.
 */
__global__ void levinLaplacian( BandedMatrix* L, float* b, float lambda, float4* im, int imH, int imW, int imPitch )
{
   const float gamma = 1e1;
   const int winRad = 1;
   //const int winSize = (2*winRad+1)*(2*winRad+1);
   
   const int u = blockIdx.x*blockDim.x + threadIdx.x;
   const int v = blockIdx.y*blockDim.y + threadIdx.y;
   
   int u1, v1, u2, v2;
   int i, j;
   int numNeighbors;
   
   float4 rgba, rgba2, white;
   // Local covariance matrix (symmetric).
   float c11, c12, c13,
              c22, c23,
                   c33;
   // Determinant of covariance matrix.
   float cdet;
   
   // Local color average.
   float4 mu;
   
   // Local inverse of covariance matrix (symmetric).
   float d11, d12, d13,
              d22, d23,
                   d33;

   // Construct local covariance matrix in the window.
   // NOTE: this will make some bad indexes into the texture.
   //       Need to find out if that is ok with texture indexing.
   c11 = c12 = c13 = c22 = c23 = c33 = 0.f;
   mu.x = mu.y = mu.z = mu.w = 0.f;
   numNeighbors = 0;
   for( v1 = v-winRad; v1 <= v+winRad; ++v1 )
   {
      for( u1 = u-winRad; u1 <= u+winRad; ++u1 )
      {
         rgba = im[u1 + v1*imPitch];
         mu.x += rgba.x; mu.y += rgba.y; mu.z += rgba.z;
         c11 += rgba.x*rgba.x;
         c12 += rgba.x*rgba.y;
         c13 += rgba.x*rgba.z;
         c22 += rgba.y*rgba.y;
         c23 += rgba.y*rgba.z;
         c33 += rgba.z*rgba.z;
         
         ++numNeighbors;
      }
   }
   mu.x /= numNeighbors; mu.y /= numNeighbors; mu.z /= numNeighbors;
   c11 = (c11+lambda)/numNeighbors - mu.x*mu.x;
   c12 = c12/numNeighbors          - mu.x*mu.y;
   c13 = c13/numNeighbors          - mu.x*mu.z;
   c22 = (c22+lambda)/numNeighbors - mu.y*mu.y;
   c23 = c23/numNeighbors          - mu.y*mu.z;
   c33 = (c33+lambda)/numNeighbors - mu.z*mu.z;
   
   // Get the inverse.
   cdet = -c11*c12*c12 +
          c11*c11*c22 -
          c13*c13*c22 +
          2*c12*c13*c23 -
          c11*c23*c23;
   d11 = (c11*c22 - c23*c23)/cdet;
   d12 = (c13*c23 - c11*c12)/cdet;
   d13 = (c12*c23 - c13*c22)/cdet;
   d22 = (c11*c11 - c13*c13)/cdet;
   d23 = (c12*c13 - c11*c23)/cdet;
   d33 = (c11*c22 - c12*c12)/cdet;
   
   for( v1 = v-winRad; v1 <= v+winRad; ++v1 )
   {
      for( u1 = u-winRad; u1 <= u+winRad; ++u1 )
      {
         i = u1 + v1*imW;
         rgba = im[u1 + v1*imPitch];
         // Get the whitened pixel
         rgba.x -= mu.x;
         rgba.y -= mu.y;
         rgba.z -= mu.z;
         white.x = d11 * rgba.x + d12 * rgba.y + d13 * rgba.z;
         white.y = d12 * rgba.x + d22 * rgba.y + d23 * rgba.z;
         white.z = d13 * rgba.x + d13 * rgba.y + d33 * rgba.z;
      }
   }
}

/*
__global__ void gradMatting( float* alpha, CompressedMatrix* L, texture<float,3,hipReadModeNormalizedFloat> const* image, texture<float,1,hipReadModeNormalizedFloat> const* trimap )
{
   extern __shared__ float sdata[];
   
   const int nthreads = blockDim.x*gridDim.x;
   const int ti = threadIdx.x;
   const int i = blockIdx.x*blockDim.x + threadIdx.x;
   
   float* b;
   float* d;
   float* e;
   float* f;
   float k;
   int N = L->rows;
   float tmp; // NOTE: somehow, this must be global or shared. It causes an error with atomicAdd otherwise.
   
   // Create the Laplacian.
   levinLaplacian( L, b, image, trimap );
   __syncthreads();
   
   // Do the gradient descent iteration.
   while( true )
   {
      // d := 2*L*alpha - b = gradient(alpha'*L*alpha - alpha'*b)
      vecScale( f, alpha, 2.0f, N );
      __syncthreads();
      csmAxpy<true,false>(d, L, f, b);
      __syncthreads();
      
      // If the gradient magnitude is small enough, we're done.
      innerProd(&tmp, d, d, N);

      __syncthreads();
      if( tmp < 1e-5 )
         break;
      
      // k := <d,b>
      innerProd(&k, d, b, N);
      __syncthreads();
      
      // e := H*d
      csmAx<true>(e, L, d);
      __syncthreads();
      
      // k -= 2*<e,alpha>
      innerProd( &tmp, e, alpha, N );
      __syncthreads();
      k -= 2*tmp;
      
      // k /= 2*<e,d>
      innerProd( &tmp, e, d, N );
      __syncthreads();
      k /= 2.0f*tmp;
      
      // alpha += k*d
      vecScale( d, d, k, N );
      vecAdd( alpha, alpha, d, N );
   }
}
*/
