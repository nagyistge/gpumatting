#include "hip/hip_runtime.h"
/* CoordMatrix.cu is part of gpumatting and is 
 * Copyright 2013 Philip G. Lee <rocketman768@gmail.com>
 * 
 * gpumatting is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 * 
 * gpumatting is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 * 
 * You should have received a copy of the GNU General Public License
 * along with gpumatting.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "CoordMatrix.h"
#include <hip/device_functions.h>

void cooInit( CoordMatrix* m, int rows, int cols, size_t length )
{
   m->rows = rows;
   m->cols = cols;
   m->nnz = 0;
   m->length = length;

   hipMalloc( &(m->i), length );
   hipMalloc( &(m->j), length );
   hipMalloc( &(m->k), length );
}

void cooRelease( CoordMatrix* m )
{
   hipFree( m->i );
   hipFree( m->j );
   hipFree( m->k );
   m->rows = 0;
   m->cols = 0;
   m->nnz = 0;
   m->length = 0;
}

void cooAppend( CoordMatrix* m, int* i, int* j, int* k, size_t length )
{
   hipMemcpy(m->i, i, length*sizeof(int), hipMemcpyHostToHost );
   hipMemcpy(m->j, j, length*sizeof(int), hipMemcpyHostToHost );
   hipMemcpy(m->k, k, length*sizeof(float), hipMemcpyHostToHost );
   m->nnz += length;
}

__global__ void cooAxpy( float* b, CoordMatrix const* a, float const* x, float const* y )
{
   // Assume blocks and grids are all 1D
   
   // Find out total number of threads N.
   int nthreads = blockDim.x*gridDim.x;
   // Find out thread index 0 <= ti < N.
   int ti = blockIdx.x*blockDim.x + threadIdx.x;
   
   // The end of the a->i array.
   int const* aiend = a->i + a->nnz;
   // The end of y.
   float const* yend = y + a->rows;
   
   int const* ai   = a->i + ti;
   int const* aj   = a->j + ti;
   float const* ak = a->k + ti;
   
   // Matrix multiplication.
   while( ai < aiend )
   {
      // ===COMPLEXITY===
      // Dereferences: 4
      // Int Adds:     4
      // Float Mults:  1
      // Float Adds:   1
      atomicAdd( b+*ai, (*ak)*x[*aj] );
      ai += nthreads;
      aj += nthreads;
      ak += nthreads;
   }
   
   // Add y to b.
   for( y += ti, b += ti; y < yend; y+=nthreads )
      *b += *y;
}
