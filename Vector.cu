#include "hip/hip_runtime.h"
/*
 * Vector.cu is part of gpumatting and Copyright Philip G. Lee <rocketman768@gmail.com> 2013
 * all rights reserved.
 */

#ifndef VECTOR_CU
#define VECTOR_CU

#include <hip/device_functions.h>

/*!
 * \brief Copy a host vector to a device vector.
 * \param leftPadding amount of 0-filled padding on the left of the vector.
 * \param rightPadding amount of 0-filled padding ont he right of the vector.
 */
void vecCopyToDevice(float** dx, float const* hx, int length, int leftPadding=0, int rightPadding=0 )
{
   hipMalloc((void**)dx, sizeof(float)*(length+leftPadding+rightPadding));
   hipDeviceSynchronize();
   hipMemset((void*)*dx, 0x00, sizeof(float)*(length+leftPadding+rightPadding));
   if( leftPadding )
      *dx += leftPadding;
   hipMemcpy((void*)*dx, (void*)hx, sizeof(float)*length, hipMemcpyHostToDevice);
}

void vecDeviceFree( float* dx, int leftPadding=0 )
{
   hipFree(dx-leftPadding);
}

__device__ void reduceSequential()
{
   extern __shared__ float sdata[];
   
   int ti = threadIdx.x;
   int stride;
   
   for( stride = blockDim.x>>1; stride > 0; stride >>= 1 )
   {
      if( ti < stride )
         sdata[ti] += sdata[ti+stride];
      __syncthreads();
   }
}

template <int blocksize>
__device__ void reduceUnrolled()
{
   extern __shared__ float sdata[];
   int ti = threadIdx.x;
   
   if( blocksize >= 1024 )
   {
      if( ti < 512 )
         sdata[ti] += sdata[ti+512];
      __syncthreads();
   }
   if( blocksize >= 512 )
   {
      if( ti < 256 )
         sdata[ti] += sdata[ti+256];
      __syncthreads();
   }
   if( blocksize >= 256 )
   {
      if( ti < 128 )
         sdata[ti] += sdata[ti+128];
      __syncthreads();
   }
   if( blocksize >= 128 )
   {
      if( ti < 64 )
         sdata[ti] += sdata[ti+64];
      __syncthreads();
   }
   
   // Since warp size is 32, these are guaranteed to happen synchronously,
   // so no explicity synching is needed.
   if( ti < 32 )
   {
      if( blocksize >= 64 )
         sdata[ti] += sdata[ti+32];
      if( blocksize >= 32 )
         sdata[ti] += sdata[ti+16];
      if( blocksize >= 16 )
         sdata[ti] += sdata[ti+8];
      if( blocksize >= 8 )
         sdata[ti] += sdata[ti+4];
      if( blocksize >= 4 )
         sdata[ti] += sdata[ti+2];
      if( blocksize >= 2 )
         sdata[ti] += sdata[ti+1];
   }
}

/*!
 * \brief Add vectors. Can be in-place.
 * 
 * Shared memory: 0
 * 
 * \param result output vector \c x + \c y. May be \c x or \c y.
 * \param x input vector
 * \param y second input vector
 * \param len number of elements in \c x, \c y, and \c result.
 */
__device__ void vecAdd( float* result, float const* x, float const* y, int len )
{
   int nthreads = blockDim.x*gridDim.x;
   int i = blockDim.x * blockIdx.x + threadIdx.x;
   int ti = threadIdx.x;
   
   while( i < len )
   {
      result[i] = x[i]+y[i];
      
      i += nthreads;
   }
}

__global__ void vecAdd_k( float* result, float const* x, float const* y, int len )
{
   vecAdd( result, x, y, len );
}

/*!
 * \brief Scale vector by a constant. Can be in-place.
 * 
 * Shared memory: 0
 * 
 * \param result output vector. May be \c x.
 * \param x input vector
 * \param s scaling factor
 * \param len number of elements in \c x and \c result.
 */
__device__ void vecScale( float* result, float const* x, float s, int len )
{
   int nthreads = blockDim.x*gridDim.x;
   int i = blockDim.x * blockIdx.x + threadIdx.x;
   int ti = threadIdx.x;
   
   while( i < len )
   {
      result[i] = s*x[i];
      i += nthreads;
   }
}

__global__ void vecScaleConst_k( float* result, float const* x, float s, int len )
{
   vecScale( result, x, s, len );
}

__global__ void vecScale_k( float* result, float const* x, float* s, int len )
{
   vecScale( result, x, *s, len );
}

/*!
 * \brief Stores inner product of \c x and \c y of length \c len in \c result.
 * 
 * Shared memory: blockDim.x*sizeof(float)
 * 
 * \param result scalar inner product
 * \param x first vector
 * \param y second vector
 * \param len length of \c x and \c y
 */
__device__ void innerProd( float* result, float const* x, float const* y, int len )
{
   extern __shared__ float sdata[];
   
   int nthreads = blockDim.x*gridDim.x;
   int i = blockDim.x * blockIdx.x + threadIdx.x;
   int ti = threadIdx.x;
   
   float* mysdata = sdata+ti;
   *mysdata = 0.f;
   
   for( ; i < len; i += nthreads )
      *mysdata += x[i]*y[i];
  
   // Doesn't work for some reason? 
   //if( i == 0 )
      *result = 0.f;
   
   // Wait for all the shared data to be fully populated.
   __syncthreads();
   
   /*
   switch( blockDim.x )
   {
      case 1024:
         reduceUnrolled<1024>();
         break;
      case 512:
         reduceUnrolled<512>();
         break;
      case 256:
         reduceUnrolled<256>();
         break;
      case 128:
         reduceUnrolled<128>();
         break;
      case 64:
         reduceUnrolled<64>();
         break;
      case 32:
         reduceUnrolled<32>();
         break;
      case 16:
         reduceUnrolled<16>();
         break;
      case 8:
         reduceUnrolled<8>();
         break;
      case 4:
         reduceUnrolled<4>();
         break;
      case 2:
         reduceUnrolled<2>();
         break;
      case 1:
         reduceUnrolled<1>();
         break;
   }
   */
   
   reduceSequential();
   
   // Need each block to contribute its final result to the global result.
   if( ti == 0 )
      atomicAdd( result, sdata[0] );
}

__global__ void innerProd_k( float* result, float const* x, float const* y, int len )
{
   innerProd( result, x, y, len );
}

#endif
