#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "Vector.cu"

int main()
{
   int N = 1e6;
   int nBlocks = 16;
   int nThreadsPerBlock = 1024;
   
   int i;
   float* hx = (float*)malloc( N * sizeof(float) );
   float* hy = (float*)malloc( N * sizeof(float) );
   float* dx;
   float* dy;
   float hresult = 0;
   float* dresult;
   
   for( i = 0; i < N; ++i )
   {
      hx[i] = 1.f;
      hy[i] = 2.f;
   }
   
   // Tell GPU what to do.
   hipMalloc( (void**)&dx, N*sizeof(float) );
   hipMalloc( (void**)&dy, N*sizeof(float) );
   hipMalloc( (void**)&dresult, 1*sizeof(float) );
   hipMemcpyAsync( (void*)dx, (void*)hx, N*sizeof(float), hipMemcpyHostToDevice );
   hipMemcpyAsync( (void*)dy, (void*)hy, N*sizeof(float), hipMemcpyHostToDevice );
   hipMemcpyAsync( (void*)dresult, (void*)&hresult, 1*sizeof(float), hipMemcpyHostToDevice );
   innerProd_k<<<nBlocks, nThreadsPerBlock, nThreadsPerBlock*sizeof(float)>>>(dresult, dx, dy, N);
   hipMemcpyAsync( (void*)&hresult, (void const*)dresult, 1*sizeof(float), hipMemcpyDeviceToHost );
   hipFree( dresult );
   hipFree( dy );
   hipFree( dx );
   
   // Wait for GPU to finish all that shit.
   hipDeviceSynchronize();
   
   if( fabs(hresult-2e6) < 1e-5 )
      printf("Test PASSED\n");
   else
   {
      printf("Test FAILED\n");
      printf("result: %.2e\n", hresult);
      return 1;
   }
   
   return 0;
}
