#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "BandedMatrix.h"
#include "BandedMatrix.cu"

int main()
{
   int N = 1e6;
   int nBlocks = 16;
   int nThreadsPerBlock = 1024;
   int sharedBytesPerBlock = 0;
   
   int i;
   
   float* hx = (float*)malloc( N * sizeof(float) );
   float* hb = (float*)malloc( N * sizeof(float) );
   BandedMatrix hA;
   
   float* dx;
   float* db;
   BandedMatrix dA;
   
   // Initialize the host's banded matrix.
   hA.nbands = 3;
   hA.rows = N;
   hA.cols = N;
   hA.a = (float*)malloc( N * hA.nbands * sizeof(float) );
   memset( hA.a, 0x00, N*hA.nbands * sizeof(float) );
   hA.bands = (int*)malloc( hA.nbands * sizeof(int) );
   hA.bands[0] = -10;
   hA.bands[1] = 0;
   hA.bands[2] = 10;
   
   for( i = 0; i < N; ++i )
   {
      hx[i] = i;
      
      if( i - 10 >= 0 )
         hA.a[i + 0*hA.rows] = 1;
      hA.a[i + 1*hA.rows] = 1;
      if( i + 10 < N )
         hA.a[i + 2*hA.rows] = 1;
   }
   
   sharedBytesPerBlock = hA.nbands * sizeof(int);
   
   //++++++++++++++++++++++++++Kernel Invocation+++++++++++++++++++++++++++++++
   hipMalloc( (void**)&dx, (N+10+10)*sizeof(float) );
   hipDeviceSynchronize();
   dx += 10;
   hipMemcpy( (void*)dx, (void*)hx, N*sizeof(float), hipMemcpyHostToDevice );

   hipMalloc( (void**)&db, N*sizeof(float) );
   
   bmCopyToDevice( &dA, &hA );

   bmAx_k<<<nBlocks, nThreadsPerBlock, sharedBytesPerBlock>>>(db, dA, dx);
   hipMemcpy( (void*)hb, (void*)db, N*sizeof(float), hipMemcpyDeviceToHost );
   
   bmDeviceFree( &dA );
   hipFree( db );
   hipFree( dx - 10 );
 
   // Wait for GPU to finish all that shit.
   hipDeviceSynchronize();
   //--------------------------------------------------------------------------
   
   
   //+++++++++++++++++++++++++++++++TEST+++++++++++++++++++++++++++++++++++++++
   bool passed = true;
   for( i = 0; i < 10; ++i )
   {
      if(fabs(hb[i] - (10.f+2*i)) > 1e-6)
      {
         printf("b[%d]=%.5e\n", i, hb[i]);
         passed = false;
         break;
      }
   }
   for( i = 10; passed && i < N-10; ++i )
   {
      if(fabs(hb[i] - (30.0+3*(i-10))) > 1e-6)
      {
         printf("b[%d]=%.5e\n", i, hb[i]);
         passed = false;
         break;
      }
   }
   for( i = N-10; passed && i < N; ++i )
   {
      if(fabs(hb[i] - (1999970.0+2*(i-(N-10)))) > 1e-6)
      {
         printf("b[%d]=%.5e\n", i, hb[i]);
         passed = false;
         break;
      }
   }
   
   if(passed)
      printf("Test PASSED\n");
   else
      printf("Test FAILED\n");
   //--------------------------------------------------------------------------
   
   free(hA.bands);
   free(hA.a);
   free(hb);
   free(hx);
   return 0;
}

