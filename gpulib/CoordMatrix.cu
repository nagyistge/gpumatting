/*
 * CoordMatrix.cu is part of gpumatting and Copyright Philip G. Lee <rocketman768@gmail.com> 2013
 * all rights reserved.
 */

#include "CoordMatrix.h"

void cmInit( CoordMatrix* m, int rows, int cols, size_t length )
{
   m->rows = rows;
   m->cols = cols;
   m->nnz = 0;
   m->length = length;

   hipMalloc( &(m->i), length );
   hipMalloc( &(m->j), length );
   hipMalloc( &(m->k), length );
}

void cmRelease( CoordMatrix* m )
{
   hipFree( m->i );
   hipFree( m->j );
   hipFree( m->k );
   m->rows = 0;
   m->cols = 0;
   m->nnz = 0;
   m->length = 0;
}
