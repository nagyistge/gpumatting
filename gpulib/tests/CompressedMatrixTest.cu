#include <stdio.h>
#include <hip/hip_runtime.h>
#include "Vector.h"
#include "CompressedMatrix.h"

int main()
{
   int N = 1e6;
   int nBlocks = 16;
   int nThreadsPerBlock = 1024;
   
   int i,col,kk;
   float* hx = (float*)malloc( N * sizeof(float) );
   float* hy = (float*)malloc( N * sizeof(float) );
   float* dx;
   float* dy;
   CompressedMatrix* da;
   float* hb = (float*)malloc( N * sizeof(float) );
   float* db;
   
   // Make x and y vectors.
   for( i = 0; i < N; ++i )
   {
      hx[i] = 1.f;
      hy[i] = 2.f;
   }

   // Make the matrix.
   CompressedMatrix* ha = (CompressedMatrix*)malloc(sizeof(CompressedMatrix));
   ha->length = 9*N;
   ha->rows = N;
   ha->cols = N;
   float* k = (float*)malloc( 9*N*sizeof(float) );
   int* j = (int*)malloc( 9*N*sizeof(int) );
   int* p = (int*)malloc( (N+1)*sizeof(int) );
   for( i = 0, kk=0; i < N; ++i )
   {
      p[i] = kk;
      for( col = i; col < N && col-i < 9; ++col, ++kk )
      {
         k[kk] = (float)kk;
         j[kk] = col;
      }
   }
   p[i] = kk;
   ha->nnz = kk;
   
   // Tell GPU what to do.
   hipMalloc( (void**)&dx, N*sizeof(float) );
   hipMalloc( (void**)&dy, N*sizeof(float) );
   hipMalloc( (void**)&db, N*sizeof(float) );
   
   hipMalloc( (void**)&da, sizeof(CompressedMatrix) );
   hipMalloc( (void**)&(da->k), 9*N*sizeof(float) );
   hipMalloc( (void**)&(da->j), 9*N*sizeof(int) );
   hipMalloc( (void**)&(da->p), (N+1)*sizeof(int) );
   
   hipMemcpyAsync( (void*)dx, (void*)hx, N*sizeof(float), hipMemcpyHostToDevice );
   hipMemcpyAsync( (void*)dy, (void*)hy, N*sizeof(float), hipMemcpyHostToDevice );
   
   // Copy the matrix over.
   hipMemcpyAsync( (void*)da, (void*)ha, sizeof(CompressedMatrix), hipMemcpyHostToDevice );
   hipMemcpyAsync( (void*)(da->k), (void*)k, 9*N*sizeof(float), hipMemcpyHostToDevice );
   hipMemcpyAsync( (void*)(da->j), (void*)j, 9*N*sizeof(int), hipMemcpyHostToDevice );
   hipMemcpyAsync( (void*)(da->p), (void*)p, (N+1)*sizeof(float), hipMemcpyHostToDevice );
   
   // Do the damn multiplication already.
   csmAxpy<<<nBlocks, nThreadsPerBlock, nThreadsPerBlock*sizeof(float)>>>(db, da, dx, dy);
   
   // Copy result vector back.
   hipMemcpyAsync( (void*)hb, (void const*)db, N*sizeof(float), hipMemcpyDeviceToHost );
   
   // Free device pointers.
   hipFree( da->p );
   hipFree( da->j );
   hipFree( da->k );
   hipFree( da );
   hipFree( db );
   hipFree( dy );
   hipFree( dx );
   
   // Wait for GPU to finish all that shit.
   hipDeviceSynchronize();
   
   // Print result
   printf("b=\n");
   for( i = 0; i < 10; ++i )
      printf("%.2e\n", hb[i]);
   printf("...\n");
   
   // Free host pointers.
   free( p );
   free( j );
   free( k );
   free( ha );
   free( hb );
   free( hy );
   free( hx );
   
   return 0;
}

