#include <stdio.h>
#include <hip/hip_runtime.h>
#include "Vector.h"

int main()
{
   int N = 1e6;
   int nBlocks = 16;
   int nThreadsPerBlock = 1024;
   
   int i;
   float* hx = (float*)malloc( N * sizeof(float) );
   float* hy = (float*)malloc( N * sizeof(float) );
   float* dx;
   float* dy;
   float hresult = 0;
   float* dresult;
   
   for( i = 0; i < N; ++i )
   {
      hx[i] = 1.f;
      hy[i] = 2.f;
   }
   
   // Tell GPU what to do.
   hipMalloc( (void**)&dx, N*sizeof(float) );
   hipMalloc( (void**)&dy, N*sizeof(float) );
   hipMalloc( (void**)&dresult, 1*sizeof(float) );
   hipMemcpyAsync( (void*)dx, (void*)hx, N*sizeof(float), hipMemcpyHostToDevice );
   hipMemcpyAsync( (void*)dy, (void*)hy, N*sizeof(float), hipMemcpyHostToDevice );
   hipMemcpyAsync( (void*)dresult, (void*)&hresult, 1*sizeof(float), hipMemcpyHostToDevice );
   innerProd<<<nBlocks, nThreadsPerBlock, nThreadsPerBlock*sizeof(float)>>>(dresult, dx, dy, N);
   hipMemcpyAsync( (void*)&hresult, (void const*)dresult, 1*sizeof(float), hipMemcpyDeviceToHost );
   hipFree( dresult );
   hipFree( dy );
   hipFree( dx );
   
   // Wait for GPU to finish all that shit.
   hipDeviceSynchronize();
   
   printf("result: %.2e\n", hresult);
   return 0;
}

