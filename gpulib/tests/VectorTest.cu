#include <stdio.h>
#include <hip/hip_runtime.h>
#include "Vector.h"

int main()
{
   int N = 1e6;
   int nBlocks = 8;
   int nThreadsPerBlock = 32;
   
   int i;
   float* hx = (float*)malloc( N * sizeof(float) );
   float* hy = (float*)malloc( N * sizeof(float) );
   float* dx;
   float* dy;
   float hresult;
   float* dresult;
   
   for( i = 0; i < N; ++i )
   {
      hx[i] = 1.f;
      hy[i] = 2.f;
   }
   
   // Tell GPU what to do.
   hipMalloc( (void**)&dx, N*sizeof(float) );
   hipMalloc( (void**)&dy, N*sizeof(float) );
   hipMalloc( (void**)&dresult, 1*sizeof(float) );
   hipMemcpy( (void*)dx, (void*)hx, N*sizeof(float), hipMemcpyHostToDevice );
   hipMemcpy( (void*)dy, (void*)hy, N*sizeof(float), hipMemcpyHostToDevice );
   innerProd<<<nBlocks, nThreadsPerBlock, nThreadsPerBlock*sizeof(float)>>>(dresult, dx, dy, N);
   hipMemcpy( (void*)&hresult, (void const*)dresult, 1*sizeof(float), hipMemcpyDeviceToHost );
   hipFree( dresult );
   hipFree( dy );
   hipFree( dx );
   
   // Wait for GPU to finish all that shit.
   hipDeviceSynchronize();
   
   printf("%.2e\n", hresult);
   return 0;
}
