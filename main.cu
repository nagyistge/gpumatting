#include "hip/hip_runtime.h"
/* main.cu is part of gpumatting and is 
 * Copyright 2013 Philip G. Lee <rocketman768@gmail.com>
 * 
 * gpumatting is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 * 
 * gpumatting is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 * 
 * You should have received a copy of the GNU General Public License
 * along with gpumatting.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include "ppm.h"
#include "BandedMatrix.h"
#include "BandedMatrix.cu"
#include "Matting.cu"
#include "Vector.cu"
#include "SLIC.h"

//! \brief Print help message and exit.
void help();
//! \brief Dump vector to stdout in %.5e format.
void dump1D( float* a, int n );
//! \brief Dump row-major matrix to stdout in %.5e format.
void dump2D( float* a, int rows, int cols, size_t pitch );
/*!
 * \brief Solve L*alpha = b by gradient descent.
 * 
 * \param alpha device vector of size L.rows padded properly to make \c L * \c alpha work.
 * \param L device banded matrix
 * \param b device vector of size L.rows
 * \param iterations number of gradient descent steps before termination
 * \param pad The size of left and right vector padding to make \c L * x work for a vector x.
 */
void gradSolve( float* alpha, BandedMatrix L, float* b, int iterations, int pad);
/*!
 * \brief Solve L*alpha = b by conjugate-gradient descent.
 * 
 * \param alpha device vector of size L.rows padded properly to make \c L * \c alpha work.
 * \param L device banded matrix
 * \param b device vector of size L.rows
 * \param pad The size of left and right vector padding to make \c L * x work for a vector x.
 * \param iterations number of steps before termination
 * \param restartInterval restart cg after this many iterations (typically about 50)
 */
void cgSolve( float* alpha, BandedMatrix L, float* b, int pad, int iterations, int restartInterval);
/*!
 * \brief Compute and display matte ground truth errors.
 *
 * \param alpha Computed alpha matte
 * \param gtAlpha Ground truth alpha matte
 * \param imW Matte width
 * \param imH Matte height
 */
void computeError( float* alpha, float* gtAlpha, int imW, int imH );

int myceildiv(int a, int b)
{
   if( a % b != 0 )
      ++a;
   return a/b;
}

int main(int argc, char* argv[])
{
   enum Solver{SOLVER_GRAD, SOLVER_CG};
   Solver solver = SOLVER_CG;
   float4* im;
   unsigned char* charIm;
   unsigned char* scribs;
   int* labels;
   //unsigned int numLabels;
   float* b;
   float* dB;
   float* alpha;
   float* dAlpha;
   int dAlpha_pad;
   float* alphaGt = 0;
   int imW, imH;
   int scribW, scribH;
   int gtW, gtH;
   int i;
   int iterations;
   clock_t beg,end;
   
   if( argc < 5 )
      help();
   
   //==================HOST DATA====================
      
   // Parse the options.
   if( strncmp(argv[1],"grad",4)==0 )
      solver = SOLVER_GRAD;
   else
      solver = SOLVER_CG;
   iterations = atoi(argv[2]);
   im = ppmread_float4( &charIm, argv[3], &imW, &imH );
   scribs = pgmread( argv[4], &scribW, &scribH );
   if( scribW != imW || scribH != imH )
   {
      fprintf(
         stderr,
         "ERROR: scribbles not the same size as the image.\n"
         "  %d x %d vs. %d x %d\n",
         scribW, scribH, imW, imH
      );
      exit(1);
   }
   if( argc > 5 )
      alphaGt = pgmread_float( argv[5], &gtW, &gtH );
   
   BandedMatrix L;
   L.rows = imW*imH;
   L.cols = L.rows;
   // Setup bands===
   L.nbands = 17;
   L.bands = (int*)malloc(17*sizeof(int));
   L.bands[8+0] = 0;
   L.bands[8+1] = 1;
   L.bands[8+2] = 2;
   L.bands[8+3] = imW;
   L.bands[8+4] = imW+1;
   L.bands[8+5] = imW+2;
   L.bands[8+6] = 2*imW;
   L.bands[8+7] = 2*imW+1;
   L.bands[8+8] = 2*imW+2;
   L.bands[8-1] = -1;
   L.bands[8-2] = -2;
   L.bands[8-3] = -imW;
   L.bands[8-4] = -(imW+1);
   L.bands[8-5] = -(imW+2);
   L.bands[8-6] = -(2*imW);
   L.bands[8-7] = -(2*imW+1);
   L.bands[8-8] = -(2*imW+2);
   // Setup nonzeros===
   L.a = (float*)malloc( L.nbands*L.rows * sizeof(float));
   memset( L.a, 0x00, L.nbands*L.rows * sizeof(float));
   L.apitch = L.rows;
   
   b = (float*)malloc( L.rows * sizeof(float) );
   alpha = (float*)malloc(L.rows * sizeof(float));
   for( i = 0; i < L.rows; ++i )
      alpha[i] = 0.5f;
   
   labels = (int*)malloc(imW*imH*sizeof(int));
   beg = clock();
   // charIm is [r,g,b,a,r,g,b,a...], but slic wants [a,r,g,b,a,r,g,b,...], so shift the
   // charIm by 1 to appease it.
   // WARNING: this may cause a segfault since it can result in a bad dereference.
   //numLabels = slicSegmentation( labels, (unsigned int*)(charIm-1), imW, imH, 100, 10.0 );
   //end = clock();
   //fprintf(stderr,"SLIC segmentation: %.2es\n", (double)(end-beg)/CLOCKS_PER_SEC);
   
   beg = clock();
   // WARNING: regularization param < 1e-3 seems to make the Laplacian unstable.
   hostLevinLaplacian(L, b, 1e-3, im, scribs, imW, imH, imW);
   end = clock();
   fprintf(stderr,"Laplacian generation: %.2es\n", (double)(end-beg)/CLOCKS_PER_SEC);
   //------------------------------------------------
   
   // Pad alpha by a multiple of 32 that is larger than (2*imW+2).
   dAlpha_pad = ((2*imW+2)/32)*32+32;
   
   //=================GPU Time=======================
   hipDeviceSetCacheConfig(hipFuncCachePreferL1);
   BandedMatrix dL;
   bmCopyToDevice( &dL, &L );
   
   hipMalloc((void**)&dB, L.rows*sizeof(float));
   hipMemcpy((void*)dB, (void*)b, L.rows*sizeof(float), hipMemcpyHostToDevice);
   
   vecCopyToDevice(&dAlpha, alpha, L.rows, dAlpha_pad, dAlpha_pad);
   
   //+++++++++++++++++++++++++++++
   switch( solver )
   {
      case SOLVER_GRAD:
         gradSolve(dAlpha, dL, dB, iterations, dAlpha_pad);
         break;
      case SOLVER_CG:
         cgSolve(dAlpha, dL, dB, dAlpha_pad, iterations, 101);
         break;
      default:
         break;
   }
   //+++++++++++++++++++++++++++++
   
   hipMemcpy( (void*)alpha, (void*)dAlpha, L.rows*sizeof(float), hipMemcpyDeviceToHost );
   
   vecDeviceFree( dAlpha, dAlpha_pad );
   hipFree(dB);
   bmDeviceFree( &dL );
   
   hipDeviceSynchronize();
   //------------------------------------------------
   
   // Print any errors
   hipError_t code = hipGetLastError(); 
   const char* error_str = hipGetErrorString(code);
   if( code )
      fprintf(stderr, "ERROR: %s\n", error_str);
   
   // Print some stats
   printf("Pitch: %lu, %lu\n", L.apitch, dL.apitch);
   printf("rows, nbands: %d, %d\n", dL.rows, dL.nbands);
   printf("Image Size: %d x %d\n", imW, imH );
   
   if(alphaGt)
      computeError(alpha, alphaGt, imW, imH);
   
   pgmwrite_float("alpha.pgm", imW, imH, alpha, "", 1);
   
   free(alpha);
   free(b);
   free(L.a);
   free(L.bands);
   free(labels);
   free(scribs);
   free(im);
   free(charIm);
   return 0;
}

void help()
{
   fprintf(
      stderr,
      "Usage: matting <solver> <iter> <image>.ppm <scribbles>.pgm [<gt>.pgm]\n"
      "  solver    - Either \"grad\" or \"cg\" for gradient/conjugate-gradient\n"
      "  iter      - Number of iterations for the solver\n"
      "  image     - An RGB image to matte\n"
      "  scribbles - Scribbles for the matte\n"
      "  gt        - Ground truth for the matte\n"
   );
   
   exit(0);
}

void dump1D( float* a, int n )
{
   int i;
   for( i = 0; i < n-1; ++i )
      printf("%.5e, ", a[i]);
   printf("%.5e\n", a[i]);
}

void dump2D( float* a, int rows, int cols, size_t pitch )
{
   int i,j;
   for( i = 0; i < rows; ++i )
   {
      for( j = 0; j < cols-1; ++j )
         printf("%.5e, ", a[j + i*pitch]);
      printf("%.5e\n", a[j + i*pitch]);
   }
}

__global__ void addScalar( float* k, float* val )
{
   *k += *val;
}

__global__ void subScalar( float* k, float* val )
{
   *k -= *val;
}

__global__ void multScalar( float* k, float* val )
{
   *k *= *val;
}

__global__ void multScalarConst( float* k, float val )
{
   *k *= val;
}

__global__ void divScalar( float* k, float* val )
{
   *k /= *val;
}

__global__ void divScalar2( float* lhs, float* num, float* den )
{
   *lhs = *num / *den;
}

void gradSolve( float* alpha, BandedMatrix L, float* b, int iterations, int pad)
{
   float* d;
   float* e;
   float* f;
   float* k;
   int N = L.rows;
   float* tmp;
   
   vecDeviceMalloc(&d, N, pad, pad);
   hipMalloc((void**)&e, N*sizeof(float));
   vecDeviceMalloc(&f, N, pad, pad);
   hipMalloc((void**)&k, 1*sizeof(float));
   hipMalloc((void**)&tmp, 1*sizeof(float));
   
   hipDeviceSynchronize();
   
   // Do the gradient descent iteration.
   while( iterations-- > 0 )
   {
      // d := L*alpha - b
      bmAxpy_k<17,false><<<16,1024>>>(d, L, alpha, b);
      
      // If the gradient magnitude is small enough, we're done.
      //innerProd(&tmp, d, d, N);
      
      // k := <d,b>
      innerProd_k<<<16,1024,1024*sizeof(float)>>>(k, d, b, N);
      
      // e := H*d
      bmAx_k<17><<<16,1024>>>(e, L, d);
      
      // k -= <e,alpha>
      innerProd_k<<<16,1024,1024*sizeof(float)>>>( tmp, e, alpha, N );
      subScalar<<<1,1>>>(k,tmp);
      
      // k /= <e,d>
      innerProd_k<<<16,1024,1024*sizeof(float)>>>( tmp, e, d, N );
      divScalar<<<1,1>>>(k, tmp);
      
      // alpha += k*d
      vecScale_k<<<16,1024>>>( d, d, k, N );
      vecAdd_k<<<16,1024>>>( alpha, alpha, d, N );
   }
   
   hipFree(tmp);
   hipFree(k);
   vecDeviceFree(f, pad);
   hipFree(e);
   vecDeviceFree(d, pad);
}

void cgSolve( float* alpha, BandedMatrix L, float* b, int pad, int iterations, int restartInterval)
{
   float* r;
   float* p;
   float* Lp;
   float* kp;
   float* k;
   int N = L.rows;
   float* rTr;
   
   // This makes the first iteration gradient descent.
   int innerIter = 0;
   
   vecDeviceMalloc(&r, N, pad, pad);
   vecDeviceMalloc(&p, N, pad, pad);
   vecDeviceMalloc(&Lp, N, pad, pad);
   vecDeviceMalloc(&kp, N, 0, 0);
   hipMalloc((void**)&k, 1*sizeof(float));
   hipMalloc((void**)&rTr, 1*sizeof(float));
   
   hipDeviceSynchronize();
   
   // Do the conjugate gradient iterations.
   while( iterations-- > 0 )
   {
      if( innerIter == 0 )
      {
         // r := L*alpha - b
         bmAxpy_k<17,false><<<16,1024>>>(r, L, alpha, b);
         // p = -r
         vecScaleConst_k<<<16,1024>>>(p, r, -1.0f, N);
         
         innerIter = restartInterval-1;
      }
      else
         --innerIter;
      
      // Lp := L*p
      bmAx_k<17><<<16,1024>>>(Lp, L, p);
      
      // k = <r,r>/<p,p>_L
      innerProd_k<<<16,1024,1024*sizeof(float)>>>(rTr, r, r, N);
      innerProd_k<<<16,1024,1024*sizeof(float)>>>(k, p, Lp, N);
      divScalar2<<<1,1>>>(k,rTr,k);
      
      // alpha += k*p
      vecScale_k<<<16,1024>>>(kp, p, k, N);
      vecAdd_k<<<16,1024>>>(alpha, alpha, kp, N);
      
      // r += k*L*p
      vecScale_k<<<16,1024>>>(Lp, Lp, k, N);
      vecAdd_k<<<16,1024>>>(r, r, Lp, N);
      
      // k = <r,r>/<r_old,r_old>
      innerProd_k<<<16,1024,1024*sizeof(float)>>>(k, r, r, N);
      divScalar<<<1,1>>>(k,rTr);
      
      // p = k*p - r;
      vecScale_k<<<16,1024>>>(kp, p, k, N);
      vecSub_k<<<16,1024>>>( p, kp, r, N );
   }
   
   hipFree(rTr);
   hipFree(k);
   vecDeviceFree(kp, 0);
   vecDeviceFree(Lp, pad);
   vecDeviceFree(p, pad);
   vecDeviceFree(r, pad);
}

void computeError( float* alpha, float* gtAlpha, int imW, int imH )
{
   double ssd = 0.0;
   int i, j;
   
   for( i = 0; i < imH; ++i )
   {
      for( j = 0; j < imW; ++j )
      {
         if( alpha[j + i*imW] > 1.0f )
            ssd += (1.0f-gtAlpha[j+i*imW])*(1.0f-gtAlpha[j+i*imW]);
         else if( alpha[j + i*imW] < 0.0f )
            ssd += gtAlpha[j+i*imW] * gtAlpha[j+i*imW];
         else
            ssd += (alpha[j+i*imW]-gtAlpha[j+i*imW])*(alpha[j+i*imW]-gtAlpha[j+i*imW]);
      }
   }
   
   ssd /= imW*imH;
   
   printf("Ground truth SSD: %.3e\n", ssd);
}
