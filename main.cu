#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include "ppm.h"
#include "BandedMatrix.h"
#include "BandedMatrix.cu"
#include "Matting.cu"
#include "Vector.cu"

//! \brief Print help message and exit.
void help();
//! \brief Dump vector to stdout in %.5e format.
void dump1D( float* a, int n );
//! \brief Dump row-major matrix to stdout in %.5e format.
void dump2D( float* a, int rows, int cols, size_t pitch );
/*!
 * \brief Solve L*alpha = b by gradient descent.
 * 
 * \param alpha device vector of size L.rows padded properly to make \c L * \c alpha work.
 * \param L device banded matrix
 * \param b device vector of size L.rows
 * \param iterations number of gradient descent steps before termination
 * \param pad The size of left and right vector padding to make \c L * x work for a vector x.
 */
void gradSolve( float* alpha, BandedMatrix L, float* b, int iterations, int pad);
/*!
 * \brief Solve L*alpha = b by conjugate-gradient descent.
 * 
 * \param alpha device vector of size L.rows padded properly to make \c L * \c alpha work.
 * \param L device banded matrix
 * \param b device vector of size L.rows
 * \param pad The size of left and right vector padding to make \c L * x work for a vector x.
 * \param iterations number of steps before termination
 * \param restartInterval restart cg after this many iterations (typically about 50)
 */
void cgSolve( float* alpha, BandedMatrix L, float* b, int pad, int iterations, int restartInterval);

int myceildiv(int a, int b)
{
   if( a % b != 0 )
      ++a;
   return a/b;
}

int main(int argc, char* argv[])
{
   float4* im;
   float4* dIm;
   unsigned char* scribs;
   float* b;
   float* dB;
   float* alpha;
   float* dAlpha;
   int dAlpha_pad;
   int imW, imH;
   int scribW, scribH;
   int i;
   clock_t beg,end;
   
   if( argc < 3 )
      help();
   
   //==================HOST DATA====================
   im = ppmread_float4( argv[1], &imW, &imH );
   scribs = pgmread( argv[2], &scribW, &scribH );
   if( scribW != imW || scribH != imH )
   {
      fprintf(
         stderr,
         "ERROR: scribbles not the same size as the image.\n"
         "  %d x %d vs. %d x %d\n",
         scribW, scribH, imW, imH
      );
      exit(1);
   }
   
   BandedMatrix L;
   L.rows = imW*imH;
   L.cols = L.rows;
   // Setup bands===
   L.nbands = 17;
   L.bands = (int*)malloc(17*sizeof(int));
   L.bands[8+0] = 0;
   L.bands[8+1] = 1;
   L.bands[8+2] = 2;
   L.bands[8+3] = imW;
   L.bands[8+4] = imW+1;
   L.bands[8+5] = imW+2;
   L.bands[8+6] = 2*imW;
   L.bands[8+7] = 2*imW+1;
   L.bands[8+8] = 2*imW+2;
   L.bands[8-1] = -1;
   L.bands[8-2] = -2;
   L.bands[8-3] = -imW;
   L.bands[8-4] = -(imW+1);
   L.bands[8-5] = -(imW+2);
   L.bands[8-6] = -(2*imW);
   L.bands[8-7] = -(2*imW+1);
   L.bands[8-8] = -(2*imW+2);
   // Setup nonzeros===
   L.a = (float*)malloc( L.nbands*L.rows * sizeof(float));
   memset( L.a, 0x00, L.nbands*L.rows * sizeof(float));
   L.apitch = L.rows;
   
   b = (float*)malloc( L.rows * sizeof(float) );
   alpha = (float*)malloc(L.rows * sizeof(float));
   for( i = 0; i < L.rows; ++i )
      alpha[i] = 0.5f;
   
   beg = clock();
   hostLevinLaplacian(L, b, 1e-2, im, scribs, imW, imH, imW);
   end = clock();
   //dump1D( b, L.rows );
   //return 0;
   //dump2D( L.a, L.nbands, L.rows, L.rows );
   fprintf(stderr,"Laplacian generation: %.2es\n", (double)(end-beg)/CLOCKS_PER_SEC);
   //------------------------------------------------
   
   // Pad alpha by a multiple of 32 that is larger than (2*imW+2).
   dAlpha_pad = ((2*imW+2)/32)*32+32;
   
   //=================GPU Time=======================
   BandedMatrix dL;
   bmCopyToDevice( &dL, &L );
   
   hipMalloc((void**)&dB, L.rows*sizeof(float));
   hipMemcpy((void*)dB, (void*)b, L.rows*sizeof(float), hipMemcpyHostToDevice);
   
   vecCopyToDevice(&dAlpha, alpha, L.rows, dAlpha_pad, dAlpha_pad);
   
   //+++++++++++++++++++++++++++++
   gradSolve(dAlpha, dL, dB, 100, dAlpha_pad);
   //+++++++++++++++++++++++++++++
   
   hipMemcpy( (void*)alpha, (void*)dAlpha, L.rows*sizeof(float), hipMemcpyDeviceToHost );
   
   vecDeviceFree( dAlpha, dAlpha_pad );
   hipFree(dB);
   bmDeviceFree( &dL );
   
   hipDeviceSynchronize();
   //------------------------------------------------
   
   // Print any errors
   hipError_t code = hipGetLastError(); 
   const char* error_str = hipGetErrorString(code);
   if( code )
      fprintf(stderr, "ERROR: %s\n", error_str);
   
   // Print some stats
   printf("Pitch: %lu, %lu\n", L.apitch, dL.apitch);
   printf("rows, nbands: %d, %d\n", dL.rows, dL.nbands);
   printf("Image Size: %d x %d\n", imW, imH );
   
   pgmwrite_float("alpha.pgm", imW, imH, alpha, "", 1);
   
   free(alpha);
   free(b);
   free(L.a);
   free(L.bands);
   free(im);
   return 0;
}

void help()
{
   fprintf(
      stderr,
      "Usage: matting <image>.ppm <scribbles>.pgm\n"
      "  image     - An RGB image to matte\n"
      "  scribbles - Scribbles for the matte\n"
   );
   
   exit(0);
}

void dump1D( float* a, int n )
{
   int i;
   for( i = 0; i < n-1; ++i )
      printf("%.5e, ", a[i]);
   printf("%.5e\n", a[i]);
}

void dump2D( float* a, int rows, int cols, size_t pitch )
{
   int i,j;
   for( i = 0; i < rows; ++i )
   {
      for( j = 0; j < cols-1; ++j )
         printf("%.5e, ", a[j + i*pitch]);
      printf("%.5e\n", a[j + i*pitch]);
   }
}

__global__ void addScalar( float* k, float* val )
{
   *k += *val;
}

__global__ void subScalar( float* k, float* val )
{
   *k -= *val;
}

__global__ void multScalar( float* k, float* val )
{
   *k *= *val;
}

__global__ void multScalarConst( float* k, float val )
{
   *k *= val;
}

__global__ void divScalar( float* k, float* val )
{
   *k /= *val;
}

__global__ void divScalar2( float* lhs, float* num, float* den )
{
   *lhs = *num / *den;
}

void gradSolve( float* alpha, BandedMatrix L, float* b, int iterations, int pad)
{
   float* d;
   float* e;
   float* f;
   float* k;
   int N = L.rows;
   float* tmp;
   
   float kDebug;
   
   vecDeviceMalloc(&d, N, pad, pad);
   hipMalloc((void**)&e, N*sizeof(float));
   vecDeviceMalloc(&f, N, pad, pad);
   hipMalloc((void**)&k, 1*sizeof(float));
   hipMalloc((void**)&tmp, 1*sizeof(float));
   
   hipDeviceSynchronize();
   
   // Do the gradient descent iteration.
   while( iterations-- > 0 )
   {
      // d := 2*L*alpha - b = gradient(alpha'*L*alpha - alpha'*b)
      vecScaleConst_k<<<16,1024>>>( f, alpha, 2.0f, N );
      bmAxpy_k<17,false><<<16,1024>>>(d, L, f, b);
      
      // If the gradient magnitude is small enough, we're done.
      //innerProd(&tmp, d, d, N);
      
      // k := <d,b>
      innerProd_k<<<16,1024,1024*sizeof(float)>>>(k, d, b, N);
      
      // e := H*d
      bmAx_k<17><<<16,1024>>>(e, L, d);
      
      // k -= 2*<e,alpha>
      innerProd_k<<<16,1024,1024*sizeof(float)>>>( tmp, e, alpha, N );
      multScalarConst<<<1,1>>>(tmp, 2.0f);
      subScalar<<<1,1>>>(k,tmp);
      
      // k /= 2*<e,d>
      innerProd_k<<<16,1024,1024*sizeof(float)>>>( tmp, e, d, N );
      multScalarConst<<<1,1>>>(tmp, 2.0f);
      divScalar<<<1,1>>>(k, tmp);
      
      // alpha += k*d
      vecScale_k<<<16,1024>>>( d, d, k, N );
      vecAdd_k<<<16,1024>>>( alpha, alpha, d, N );
   }
   
   hipFree(tmp);
   hipFree(k);
   vecDeviceFree(f, pad);
   hipFree(e);
   vecDeviceFree(d, pad);
}

void cgSolve( float* alpha, BandedMatrix L, float* b, int pad, int iterations, int restartInterval)
{
   float* r;
   float* p;
   float* Lp;
   float* kp;
   float* k;
   int N = L.rows;
   float* tmp;
   
   int innerIter = restartInterval;
   
   vecDeviceMalloc(&r, N, pad, pad);
   vecDeviceMalloc(&p, N, pad, pad);
   vecDeviceMalloc(&Lp, N, pad, pad);
   vecDeviceMalloc(&kp, N, 0, 0);
   hipMalloc((void**)&k, 1*sizeof(float));
   hipMalloc((void**)&tmp, 1*sizeof(float));
   
   hipDeviceSynchronize();
   
   // r := L*alpha - b
   bmAxpy_k<17,false><<<16,1024>>>(r, L, alpha, b);
   // p = -r
   vecScaleConst_k<<<16,1024>>>(p, r, -1.0f, N);
   
   // Do the conjugate gradient iterations.
   while( iterations-- > 0 )
   {
      if( innerIter == 0 )
      {
         // r := L*alpha - b
         bmAxpy_k<17,false><<<16,1024>>>(r, L, alpha, b);
         // p = -r
         vecScaleConst_k<<<16,1024>>>(p, r, -1.0f, N);
         
         innerIter = restartInterval-1;
      }
      else
         --innerIter;
      
      // Lp := L*p
      bmAx_k<17><<<16,1024>>>(Lp, L, p);
      
      // k = <r,r>/<p,p>_L
      innerProd_k<<<16,1024,1024*sizeof(float)>>>(tmp, r, r, N);
      innerProd_k<<<16,1024,1024*sizeof(float)>>>(k, p, Lp, N);
      divScalar2<<<1,1>>>(k,tmp,k);
      
      // alpha += k*p
      vecScale_k<<<16,1024>>>(kp, p, k, N);
      vecAdd_k<<<16,1024>>>(alpha, alpha, kp, N);
      
      // r += k*L*p
      vecScale_k<<<16,1024>>>(Lp, Lp, k, N);
      vecAdd_k<<<16,1024>>>(r, r, Lp, N);
      
      // k = <r,r>/<r_old,r_old>
      innerProd_k<<<16,1024,1024*sizeof(float)>>>(k, r, r, N);
      divScalar<<<1,1>>>(k,tmp);
      
      // p = k*p - r;
      vecSub_k<<<16,1024>>>( p, kp, r, N );
   }
   
   hipFree(tmp);
   hipFree(k);
   vecDeviceFree(p, pad);
   vecDeviceFree(r, pad);
}
