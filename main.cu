#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "ppm.h"
#include "Matting.cu"

texture<float4,hipTextureType2D,hipReadModeElementType> levinLaplacian_image;
texture<float,hipTextureType2D,hipReadModeElementType> levinLaplacian_trimap;

void help();

int main(int argc, char* argv[])
{
   float* im;
   int imW, imH;
   
   if( argc < 2 )
      help();
   
   im = ppmread_float( argv[1], &imW, &imH );
   
   free(im);
   return 0;
}

void help()
{
   fprintf(
      stderr,
      "Usage: matting <image>.ppm\n"
      "  image - An RGB image to matte\n"
   );
   
   exit(0);
}
